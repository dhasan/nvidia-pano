
#include <hip/hip_runtime.h>
#include 	<fcntl.h>
#include 	<stdio.h>
#define 	OUT_X				(3600)
#define 	OUT_Y				(1800)

#define 	SOURCE_Y 			(1200)
#define 	SOURCE_X 			(1200)

#define DEST_X 	(640)
#define DEST_Y	(640)

#define DEST_RATIO ((float)DEST_X)/((float)DEST_Y)

#define ANGLE_PHI	(0)
#define ANGLE_THETA	(0)

#define FOV_X		(90)
#define FOV_Y		(90)

#define RADIUS		(1)
//#define RADIUS		(OUT_X/(2*datum::pi))

unsigned int *sdata[6];//[1200][1200];

#define MAX(a,b) (a>b)?a:b
#define MIN(a,b) (a<b)?a:b

#ifndef __CUDACC__ 
struct float4 {
	float x;
	float y;
	float z;
	float w;
};

struct float3 {
	float x;
	float y;
	float z;

};

struct int4 {
	 unsigned int x;
	 unsigned int y;
	 unsigned int z;
	 unsigned int w;
};
#else
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
#endif

# define M_PI           3.14159265358979323846

void mul4x4x4(float *a, float *b, float *out){
	out[0] = a[0]*b[0] + a[1]*b[4] + a[2]*b[8] + a[3]*b[12];
	out[1] = a[0]*b[1] + a[1]*b[5] + a[2]*b[9] + a[3]*b[13];
	out[2] = a[0]*b[2] + a[1]*b[6] + a[2]*b[10] + a[3]*b[14];
	out[3] = a[0]*b[3] + a[1]*b[7] + a[2]*b[11] + a[3]*b[15];


	out[4] = a[4]*b[0] + a[5]*b[4] + a[6]*b[8] + a[7]*b[12];
	out[5] = a[4]*b[1] + a[5]*b[5] + a[6]*b[9] + a[7]*b[13];
	out[6] = a[4]*b[2] + a[5]*b[6] + a[6]*b[10] + a[7]*b[14];
	out[7] = a[4]*b[3] + a[5]*b[7] + a[6]*b[11] + a[7]*b[15];

	out[8] = a[8]*b[0] + a[9]*b[4] + a[10]*b[8] + a[11]*b[12];
	out[9] = a[8]*b[1] + a[9]*b[5] + a[10]*b[9] + a[11]*b[13];
	out[10] = a[8]*b[2] + a[9]*b[6] + a[10]*b[10] + a[11]*b[14];
	out[11] = a[8]*b[3] + a[9]*b[7] + a[10]*b[11] + a[11]*b[15];

	out[12] = a[12]*b[0] + a[13]*b[4] + a[14]*b[8] + a[15]*b[12];
	out[13] = a[12]*b[1] + a[13]*b[5] + a[14]*b[9] + a[15]*b[13];
	out[14] = a[12]*b[2] + a[13]*b[6] + a[14]*b[10] + a[15]*b[14];
	out[15] = a[12]*b[3] + a[13]*b[7] + a[14]*b[11] + a[15]*b[15];

}



__device__ void mul4x4x1(double *a, double *b, double *out){
	out[0] = a[0]*b[0] + a[1]*b[1] + a[2]*b[2] + a[3]*b[3];
	out[1] = a[4]*b[0] + a[5]*b[1] + a[6]*b[2] + a[7]*b[3];
	out[2] = a[8]*b[0] + a[9]*b[1] + a[10]*b[2] + a[11]*b[3];
	out[3] = a[12]*b[0] + a[13]*b[1] + a[14]*b[2] + a[15]*b[3];
}



__device__ void trans4x4(double *data, double *out){
	out[0] = data[0];
	out[1] = data[4];
	out[2] = data[8];
	out[3] = data[12];

	out[4] = data[1];
	out[5] = data[5];
	out[6] = data[9];
	out[7] = data[13];

	out[8] = data[2];
	out[9] = data[6];
	out[10]= data[10];
	out[11] = data[14];

	out[12] = data[3];
	out[13] = data[7];
	out[14] = data[11];
	out[15] = data[15];

}

__device__
double det4x4(double *a){
	double det;

	det = a[0]*a[5]*a[10]*a[15] + a[0]*a[6]*a[11]*a[13] + a[0]*a[7]*a[9]*a[14]
		   +a[1]*a[4]*a[11]*a[14] + a[1]*a[6]*a[8]*a[15] + a[1]*a[7]*a[10]*a[12]
		   +a[2]*a[4]*a[9]*a[15] + a[2]*a[5]*a[11]*a[12] + a[2]*a[7]*a[8]*a[13]
		   +a[3]*a[4]*a[10]*a[13] + a[3]*a[5]*a[8]*a[14] + a[3]*a[6]*a[9]*a[12]
		   -a[0]*a[5]*a[11]*a[14] - a[0]*a[6]*a[9]*a[15] - a[0]*a[7]*a[10]*a[13]
		   -a[1]*a[4]*a[10]*a[15] - a[1]*a[6]*a[11]*a[12] - a[1]*a[7]*a[8]*a[14]
		   -a[2]*a[4]*a[11]*a[13] - a[2]*a[5]*a[8]*a[15] - a[2]*a[7]*a[9]*a[12]
		   -a[3]*a[4]*a[9]*a[14] - a[3]*a[5]*a[10]*a[12] - a[3]*a[6]*a[8]*a[13];
	return det;
}

__device__ bool inverse4x4(double *a, double *b){
	double det = det4x4(a);

	if (det==0.0f){
		return false;
	}

	b[0] = (a[5]*a[10]*a[15] + a[6]*a[11]*a[13] + a[7]*a[9]*a[14]  -
			a[5]*a[11]*a[14] - a[6]*a[9]*a[15]  - a[7]*a[10]*a[13])/det;
	b[1] = (a[1]*a[11]*a[14] + a[2]*a[9]*a[15]  + a[3]*a[10]*a[13] -
			a[1]*a[10]*a[15] - a[2]*a[11]*a[13] - a[3]*a[9]*a[14])/det;
	b[2] = (a[1]*a[6]*a[15]  + a[2]*a[7]*a[13]  + a[3]*a[5]*a[14]  -
			a[1]*a[7]*a[14]  - a[2]*a[5]*a[15]  - a[3]*a[6]*a[13])/det;
	b[3] = (a[1]*a[7]*a[10]  + a[2]*a[5]*a[11]  + a[3]*a[6]*a[9]   -
			a[1]*a[6]*a[11]  - a[2]*a[7]*a[9]   - a[3]*a[5]*a[10])/det;
	b[4] = (a[4]*a[11]*a[14] + a[6]*a[8]*a[15]  + a[7]*a[10]*a[12] -
			a[4]*a[10]*a[15] - a[6]*a[11]*a[12] - a[7]*a[8]*a[14])/det;
	b[5] = (a[0]*a[10]*a[15] + a[2]*a[11]*a[12] + a[3]*a[8]*a[14]  -
			a[0]*a[11]*a[14] - a[2]*a[8]*a[15]  - a[3]*a[10]*a[12])/det;
	b[6] = (a[0]*a[7]*a[14]  + a[2]*a[4]*a[15]  + a[3]*a[6]*a[12]  -
			a[0]*a[6]*a[15]  - a[2]*a[7]*a[12]  - a[3]*a[4]*a[14])/det;
	b[7] = (a[0]*a[6]*a[11]  + a[2]*a[7]*a[8]   + a[3]*a[4]*a[10]  -
			a[0]*a[7]*a[10]  - a[2]*a[4]*a[11]  - a[3]*a[6]*a[8])/det;
	b[8] = (a[4]*a[9]*a[15]  + a[5]*a[11]*a[12] + a[7]*a[8]*a[13] -
			a[4]*a[11]*a[13] - a[5]*a[8]*a[15]  - a[7]*a[9]*a[12])/det;
	b[9] = (a[0]*a[11]*a[13] + a[1]*a[8]*a[15]  + a[3]*a[9]*a[12] -
			a[0]*a[9]*a[15]  - a[1]*a[11]*a[12] - a[3]*a[8]*a[13])/det;
	b[10]= (a[0]*a[5]*a[15]  + a[1]*a[7]*a[12]  + a[3]*a[4]*a[13] -
			a[0]*a[7]*a[13]  - a[1]*a[4]*a[15]  - a[3]*a[5]*a[12])/det;
	b[11]= (a[0]*a[7]*a[9]   + a[1]*a[4]*a[11]  + a[3]*a[5]*a[8]  -
			a[0]*a[5]*a[11]  - a[1]*a[7]*a[8]   - a[3]*a[4]*a[9])/det;
	b[12]= (a[4]*a[10]*a[13] + a[5]*a[8]*a[14]  + a[6]*a[9]*a[12] -
			a[4]*a[9]*a[14]  - a[5]*a[10]*a[12] - a[6]*a[8]*a[13])/det;
	b[13]= (a[0]*a[9]*a[14]  + a[1]*a[10]*a[12] + a[2]*a[8]*a[13] -
			a[0]*a[10]*a[13] - a[1]*a[8]*a[14]  - a[2]*a[9]*a[12])/det;
	b[14]= (a[0]*a[6]*a[13]  + a[1]*a[4]*a[14]  + a[2]*a[5]*a[12] -
			a[0]*a[5]*a[14]  - a[1]*a[6]*a[12]  - a[2]*a[4]*a[13])/det;
	b[15]= (a[0]*a[5]*a[10]  + a[1]*a[6]*a[8]   + a[2]*a[4]*a[9]  -
			a[0]*a[6]*a[9]   - a[1]*a[4]*a[10]  - a[2]*a[5]*a[8])/det;
	return true;
}

float det3x3(float *data){

	float p1 = *(data + 0*3 + 0) * *(data + 1*3 + 1) * *(data + 2*3 + 2);
	float p2 = *(data + 0*3 + 1) * *(data + 1*3 + 2) * *(data + 2*3 + 0);
	float p3 = *(data + 1*3 + 0) * *(data + 2*3 + 1) * *(data + 0*3 + 2);

	float n1 = *(data + 0*3 + 2) * *(data + 1*3 + 1) * *(data + 2*3 + 0);
	float n2 = *(data + 1*3 + 0) * *(data + 0*3 + 1) * *(data + 2*3 + 2);
	float n3 = *(data + 2*3 + 1) * *(data + 1*3 + 2) * *(data + 0*3 + 0);

	return p1+p2+p3-n1-n2-n3;

}

float det2x2(float *data){
	float p1 = *(data + 0*2 + 0) * *(data + 1*2 + 1);
	float n1 = *(data + 0*2 + 2) * *(data + 1*2 + 1);

	return p1-n1;
}


float det2x2args(float a11, float a12, float a21, float a22){
	return a11*a22 - a12*a21;
}

float inverse3x3(float *data, float *out){
	float det = det3x3(data);
	if (det==0)
		return false;
	out[0] =  1*det2x2args(data[4], data[5], data[7], data[8])/det;
	out[1] = -1*det2x2args(data[1], data[2], data[7], data[8])/det;
	out[2] =  1*det2x2args(data[1], data[2], data[4], data[5])/det;
	out[3] = -1*det2x2args(data[3], data[5], data[6], data[8])/det;
	out[4] =  1*det2x2args(data[0], data[2], data[6], data[8])/det;
	out[5] = -1*det2x2args(data[0], data[2], data[3], data[5])/det;
	out[6] =  1*det2x2args(data[3], data[4], data[6], data[7])/det;
	out[7] = -1*det2x2args(data[0], data[1], data[6], data[7])/det;
	out[8] =  1*det2x2args(data[0], data[1], data[3], data[4])/det;

	return true;

}

void mul3x3x3(float *a, float *b, float *out){
	 out[0] = a[0]*b[0] + a[1]*b[3] + a[2]*b[6];
	 out[1] = a[0]*b[1] + a[1]*b[4] + a[2]*b[7];
	 out[2] = a[0]*b[2] + a[1]*b[5] + a[2]*b[8];

	 out[3] = a[3]*b[0] + a[4]*b[3] + a[5]*b[6];
	 out[4] = a[3]*b[1] + a[4]*b[4] + a[5]*b[7];
	 out[5] = a[3]*b[2] + a[4]*b[5] + a[5]*b[8];

	 out[6] = a[6]*b[0] + a[7]*b[3] + a[8]*b[6];
	 out[7] = a[6]*b[1] + a[7]*b[4] + a[8]*b[7];
	 out[8] = a[6]*b[2] + a[7]*b[5] + a[8]*b[8];
}

__device__ void mul3x3x1(float *a, float *b, float *out){

	out[0] = a[0]*b[0] + a[1]*b[1] + a[2]*b[2];
	out[1] = a[3]*b[0] + a[4]*b[1] + a[5]*b[2];
	out[2] = a[6]*b[0] + a[7]*b[1] + a[8]*b[2];

}

void mul3x3x1h(float *a, float *b, float *out){

	out[0] = a[0]*b[0] + a[1]*b[1] + a[2]*b[2];
	out[1] = a[3]*b[0] + a[4]*b[1] + a[5]*b[2];
	out[2] = a[6]*b[0] + a[7]*b[1] + a[8]*b[2];

}

void trans3x3(float *data, float *out){
	out[0] = data[0];
	out[1] = data[3];
	out[2] = data[6];
	out[3] = data[1];
	out[4] = data[4];
	out[5] = data[7];
	out[6] = data[2];
	out[7] = data[5];
	out[8] = data[8];

}


__device__ float phi_to_j(float phi){
	float j = (float)(((OUT_Y-1)*phi  +  (OUT_Y-1)*0)/M_PI);
	return j;
}

__device__ float theta_to_i(float theta){
	float i = (float)(((((OUT_X-1)*theta)) + ((OUT_X-1)*0))/(2*M_PI));
	return i;
}

float deg_to_rad(float deg){
	float rad = deg*M_PI/180;
	return rad;	
}

float rad_to_deg(float rad){
	float deg = rad*180/M_PI;
	return deg;	
}


void sphere_to_cart(float3 *sph, float3 *cart){

	float x = sph->z*cos(sph->x)*sin(sph->y);
	float y = sph->z*sin(sph->x)*sin(sph->y);
	float z = sph->z*cos(sph->y);

	cart->x = x;
	cart->y = y;
	cart->z = z;
}

__device__ void cart_to_sphere(float3 *cart, float3 *sph){
	float theta;
	float phi;
	float r = sqrt((cart->x*cart->x) + (cart->y*cart->y) + (cart->z*cart->z));
	
	if (cart->x==0) 
		if (cart->y < 0)
			theta = -M_PI/2;
		else
			theta = M_PI/2;
	else
		theta = atan(cart->y/cart->x);

	phi = acos(cart->z/r);

	if (cart->x<0)
		phi*=-1;


	sph->x = theta;
	sph->y = phi;
	sph->z = r;
}

float distance(float3 *p1, float3 *p2){

	float x = p2->x - p1->x;
	float y = p2->y - p1->y;
	float z = p2->z - p1->z;

	float r = sqrt((x*x) + (y*y) + (z*z));

	return r;
}


struct plane {


	float3 dots[4];

};

int4 *xymap;
unsigned int *plane;
float4 *bmapg;

void create_out_plane(float *coord, float fov, float ratio){

	float3 cart_c;

	float3 cart_1,cart_2,cart_3,cart_4;
	float3 sph_t;

	float phi_c = deg_to_rad(ANGLE_PHI);
	float theta_c = deg_to_rad(ANGLE_THETA);



	float fov2 = fov/2.0;

	float phi_1 =		phi_c 	- 	fov2;
	float theta_1 = 	theta_c	+	fov2;

	float phi_2 = phi_c - fov2;
	float theta_2 = theta_c - fov2;

	float phi_3 = phi_c + fov2;
	float theta_3 = theta_c - fov2;

	float phi_4 = phi_c + fov2;
	float theta_4 = theta_c + fov2;

	sph_t.x = theta_1;
	sph_t.y = phi_1;
	sph_t.z = RADIUS;//OUT_X/(2*datum::pi);
	if (phi_1<0){
		phi_1 *= -1;
		if (theta_1<M_PI)
			theta_1 +=M_PI;
		else
			theta_1 -=M_PI;
	}

	if (theta_1<0){
		theta_1 = 2*M_PI + theta_1;
	}

	sphere_to_cart(&sph_t, &cart_1);
	printf("theta1: %f phi1 %f\n",rad_to_deg(theta_1), rad_to_deg(phi_1) );

	sph_t.x = theta_2;
	sph_t.y = phi_2;
	sph_t.z = RADIUS;//OUT_X/(2*datum::pi);
	if (phi_2<0){
		phi_2 *= -1;
		if (theta_2<M_PI)
			theta_2 +=M_PI;
		else
			theta_2 -=M_PI;
	}

	if (theta_2<0){
		theta_2 = 2*M_PI + theta_2;
	}


	sphere_to_cart(&sph_t, &cart_2);
	printf("theta2: %f phi2 %f\n",rad_to_deg(theta_2), rad_to_deg(phi_2));

	sph_t.x = theta_3;
	sph_t.y = phi_3;
	sph_t.z = RADIUS;//OUT_X/(2*datum::pi);

	if (phi_3<0){
		phi_3 *= -1;
		if (theta_3<M_PI)
			theta_3 +=M_PI;
		else
			theta_3 -=M_PI;
	}

	if (theta_3<0){
		theta_3 = 2*M_PI + theta_3;
	}


	sphere_to_cart(&sph_t, &cart_3);
	printf("theta3: %f phi3 %f\n",rad_to_deg(theta_3), rad_to_deg(phi_3) );

	sph_t.x = theta_4;
	sph_t.y = phi_4;
	sph_t.z = RADIUS;//OUT_X/(2*datum::pi);

	if (phi_4<0){
		phi_4 *= -1;
		if (theta_4<M_PI)
			theta_4 +=M_PI;
		else
			theta_4 -=M_PI;
	}

	if (theta_4<0){
		theta_4 = 2*M_PI + theta_4;
	}

	sphere_to_cart(&sph_t, &cart_4);
	printf("theta4: %f phi4 %f\n",rad_to_deg(theta_4), rad_to_deg(phi_4) );

	cart_c.x = (cart_1.x + cart_3.x)/2;
	cart_c.y = (cart_1.y + cart_3.y)/2;
	cart_c.z = (cart_1.z + cart_3.z)/2;

	printf("p1 x: %f, y: %f, z: %f\n",cart_1.x,cart_1.y,cart_1.z );
	printf("p2 x: %f, y: %f, z: %f\n",cart_2.x,cart_2.y,cart_2.z );
	printf("p3 x: %f, y: %f, z: %f\n",cart_3.x,cart_3.y,cart_3.z );
	printf("p4 x: %f, y: %f, z: %f\n",cart_4.x,cart_4.y,cart_4.z );
	printf("center x: %f y: %f z: %f\n",cart_c.x, cart_c.y, cart_c.z );
	coord[0]=cart_2.x;coord[1]=cart_2.y;coord[2]=cart_2.z;
	coord[3]=cart_4.x;coord[4]=cart_4.y;coord[5]=cart_4.z;
	coord[6]=cart_3.x;coord[7]=cart_3.y;coord[8]=cart_3.z;
}

void create_project_matrix(float *outplane, float *inputplane, float *pmatrix){
//	int i,j;
	
    float pa[9];// = mat(3,3);
    float p1[3];
   
    pa[0] = inputplane[0]; 					pa[1] = inputplane[1]; 					pa[2] = 1;
    pa[3] = inputplane[0] + inputplane[2]; 	pa[4] = inputplane[1]; 					pa[5] = 1;
    pa[6] = inputplane[0] + inputplane[2]; 	pa[7] = inputplane[1] + inputplane[3]; 	pa[8] = 1;
  
    p1[0] = outplane[0];
    p1[1] = outplane[3];
    p1[2] = outplane[6];

    float l1[3];
    float invpa[9];
    inverse3x3(pa, invpa);
    mul3x3x1h(invpa, p1, l1);

    pmatrix[0] = l1[0];
    pmatrix[1] = l1[1];
    pmatrix[2] = l1[2];

    p1[0] = outplane[1];
    p1[1] = outplane[4];
    p1[2] = outplane[7];

    mul3x3x1h(invpa, p1, l1);

    pmatrix[3] = l1[0];
    pmatrix[4] = l1[1];
    pmatrix[5] = l1[2];

    p1[0] = outplane[2];
    p1[1] = outplane[5];
    p1[2] = outplane[8];

    mul3x3x1h(invpa, p1, l1);

    pmatrix[6] = l1[0];
    pmatrix[7] = l1[1];
    pmatrix[8] = l1[2];
}

void create_rotate_matrix(float theta, float phi, float *rmatrix){

    float fa[9];
    float fb[9];

    fb[0] = 1; fb[1] = 0; 			fb[2] = 0;
    fb[3] = 0; fb[4] = cos(phi);	fb[5] = -sin(phi);
    fb[6] = 0; fb[7] = sin(phi);	fb[8] = cos(phi);

    fa[0] = cos(theta); fa[1] = -sin(theta); 	fa[2] = 0;
    fa[3] = sin(theta); fa[4] = cos(theta); 	fa[5] = 0;
    fa[6] = 0; 			fa[7] = 0;				fa[8] = 1;

 	
 	//rmatrix = fa * fb;
    mul3x3x3(fa,fb,rmatrix);
}


__device__ unsigned int argb_interpolate(struct float4 *gvec, unsigned int q1, unsigned int q2, unsigned int q3, unsigned int q4 ){

	float4 vec;
	vec.x = gvec->x;
	vec.y = gvec->y;
	vec.z = gvec->z;
	vec.w = gvec->w;

	unsigned char a1 = (unsigned char)((q1 & 0xFF000000) >> 24);
	unsigned char a2 = (unsigned char)((q2 & 0xFF000000) >> 24);
	unsigned char a3 = (unsigned char)((q3 & 0xFF000000) >> 24);
	unsigned char a4 = (unsigned char)((q4 & 0xFF000000) >> 24);

	unsigned char r1 = (unsigned char)((q1 & 0x00FF0000) >> 16);
	unsigned char r2 = (unsigned char)((q2 & 0x00FF0000) >> 16);
	unsigned char r3 = (unsigned char)((q3 & 0x00FF0000) >> 16);
	unsigned char r4 = (unsigned char)((q4 & 0x00FF0000) >> 16);

	unsigned char g1 = (unsigned char)((q1 & 0x0000FF00) >> 8);
	unsigned char g2 = (unsigned char)((q2 & 0x0000FF00) >> 8);
	unsigned char g3 = (unsigned char)((q3 & 0x0000FF00) >> 8);
	unsigned char g4 = (unsigned char)((q4 & 0x0000FF00) >> 8);

	unsigned char b1 = (unsigned char)((q1 & 0x000000FF) >> 0);
	unsigned char b2 = (unsigned char)((q2 & 0x000000FF) >> 0);
	unsigned char b3 = (unsigned char)((q3 & 0x000000FF) >> 0);
	unsigned char b4 = (unsigned char)((q4 & 0x000000FF) >> 0);

	unsigned char a = 	floor((vec.x * (float)a1 +
						vec.y * (float)a2 +
						vec.z * (float)a3 +
						vec.w * (float)a4));

	unsigned char r = 	floor((vec.x * (float)r1 +
						vec.y * (float)r2 +
						vec.z * (float)r3 +
						vec.w * (float)r4));

	unsigned char g = 	floor((vec.x * (float)g1 +
						vec.y * (float)g2 +
						vec.z * (float)g3 +
						vec.w * (float)g4));

	unsigned char b = 	floor((vec.x * (float)b1 +
						vec.y * (float)b2 +
						vec.z * (float)b3 +
						vec.w * (float)b4));

	unsigned int ret = (a << 24) | (r << 16) | (g << 8) | (b << 0);
	return ret;
}

__device__ unsigned int interpolate(float x, float y, unsigned int q1, unsigned int q2, unsigned int q3, unsigned int q4){
	double nv_bicubic[16];
	double nv_vals[4], nv_b[4];

	double nv_inv[16];
	double nv_transp[16];

	float4 bmap;
	unsigned int val;//,x1,y1,x2,y2;

	nv_bicubic[0] = 1.0f;
	nv_bicubic[1] = floor(x);
	nv_bicubic[2] = floor(y);
	nv_bicubic[3] = floor(x) * floor(y);
	nv_bicubic[4] = 1.0f;
	nv_bicubic[5] = floor(x);
	nv_bicubic[6] = ceil(y);
	nv_bicubic[7] = floor(x) * ceil(y);
	nv_bicubic[8] = 1.0f;
	nv_bicubic[9] = ceil(x);
	nv_bicubic[10] = floor(y);
	nv_bicubic[11] = ceil(x) * floor(y);
	nv_bicubic[12] = 1.0f;
	nv_bicubic[13] = ceil(x);
	nv_bicubic[14] = ceil(y);
	nv_bicubic[15] = ceil(x) * ceil(y);

	nv_vals[0] = 1.0f;
	nv_vals[1] = x;
	nv_vals[2] = y;
	nv_vals[3] = x*y;

	if(inverse4x4(nv_bicubic, nv_inv)){
		trans4x4(nv_inv, nv_transp);
		mul4x4x1(nv_transp, nv_vals, nv_b);
		bmap.x = nv_b[0];
		bmap.y = nv_b[1];
		bmap.z = nv_b[2];
		bmap.w = nv_b[3];
	}else{
		bmap.x = 0.25f;
		bmap.y = 0.25f;
		bmap.z = 0.25f;
		bmap.w = 0.25f;
	}

	val =  argb_interpolate(&bmap, q1,q2,q3,q4); 

	return val;
}	
__device__ unsigned int dotsmultiply(int4 *xymappt, float4 *bmappt, unsigned int **sources, int y, int x){

	xymappt += (y*OUT_X + x);

	unsigned int sid = xymappt->x >> 16;
	unsigned int *sdatapt = sources[sid];//&sdata[sid][0][0];

	unsigned int x1 = xymappt->x & 0x0000FFFF;
	unsigned int x2 = xymappt->y;
	unsigned int y1 = xymappt->z;
	unsigned int y2 = xymappt->w;

	unsigned int q =argb_interpolate(bmappt + y*OUT_X + x, 	
									 *(sdatapt + SOURCE_X*y1 + x1),
									 *(sdatapt + SOURCE_X*y2 + x1), 
									 *(sdatapt + SOURCE_X*y1 + x2),
									 *(sdatapt + SOURCE_X*y2 + x2)	); 

	return q;
}

__global__ void create_pano(float *dev_wm, int4 *dev_xymap, float4 *dev_bmap, 	unsigned int *dev_source0,
														unsigned int *dev_source1,
														unsigned int *dev_source2,
														unsigned int *dev_source3,
														unsigned int *dev_source4,
														unsigned int *dev_source5,
														unsigned int *dev_plane){

	float nv_invec[3];
	float nv_outvec[3];
	unsigned int *sources[6];
	float3 cr,sp;
	float jff, iff;

	sources[0] = dev_source0;
	sources[1] = dev_source1;
	sources[2] = dev_source2;
	sources[3] = dev_source3;
	sources[4] = dev_source4;
	sources[5] = dev_source5;

	int jj = blockIdx.y * blockDim.y + threadIdx.y;
    int ii = blockIdx.x * blockDim.x + threadIdx.x;

	nv_invec[0] = (float)ii;
	nv_invec[1] = (float)jj;
	nv_invec[2] = (float)1;
	
	mul3x3x1(dev_wm, nv_invec, nv_outvec);

	cr.x = nv_outvec[0];
	cr.y = nv_outvec[1];
	cr.z = nv_outvec[2];

	cart_to_sphere(&cr, &sp);
	if (sp.y<0){
		sp.y *= -1;
		if (sp.x<M_PI)
			sp.x +=M_PI;
		else
			sp.x -=M_PI;
	}else if (sp.y>M_PI){
		sp.y = M_PI - (sp.y - M_PI);
		if (sp.x<M_PI)
			sp.x +=M_PI;
		else
			sp.x -=M_PI;
	}
	if (sp.x<0){
		sp.x = (2*M_PI) + sp.x;
	}else if (sp.x>(2*M_PI))
		sp.x = sp.x - (2*M_PI);
		jff = phi_to_j(sp.y);
		iff = theta_to_i(sp.x);

		unsigned int q1 = dotsmultiply(dev_xymap, dev_bmap, sources, floor(jff), floor(iff));
		unsigned int q2 = dotsmultiply(dev_xymap, dev_bmap, sources, ceil(jff), floor(iff));
		unsigned int q3 = dotsmultiply(dev_xymap, dev_bmap, sources, floor(jff), ceil(iff));
		unsigned int q4 = dotsmultiply(dev_xymap, dev_bmap, sources, ceil(jff), ceil(iff));

		*(dev_plane + jj*DEST_X + ii) = interpolate(iff, jff, q1,q2,q3,q4 );

}

int main(){

    int i;

	float outplane[9];
	float pmatrix[9];
	float rmatrix[9];
	float nv_wm[9];
	float inputplane[4];
	
	create_out_plane(outplane, deg_to_rad(120), DEST_RATIO);

	inputplane[0] = 0;
	inputplane[1] = 0;
	inputplane[2] = DEST_X;
	inputplane[3] = DEST_Y;

	create_project_matrix(outplane, inputplane, pmatrix);
								//theta 		//phi
	create_rotate_matrix(deg_to_rad(0), deg_to_rad(90), rmatrix);

	mul3x3x3(rmatrix,pmatrix, nv_wm);

    FILE *xymapfd = fopen("xymap.bin", "rb");
    if (xymapfd == NULL){
    	printf("can't open xymap\n");
    	exit(1);
    }

    FILE *bmapfd = fopen("bmap.bin", "rb");
    if (bmapfd==NULL){
    	printf("can't open bmap\n");
    	exit(1);
    }
// Sources
    FILE *frfd = fopen("./cube/front.rgba", "rb");
    if (frfd==NULL){
    	printf("can't open front\n");
    	exit(1);
    }

    FILE *leftfd = fopen("./cube/left.rgba", "rb");
    if (leftfd==NULL){
    	printf("can't open left\n");
    	exit(1);
    }

    FILE *rightfd = fopen("./cube/right.rgba", "rb");
    if (rightfd==NULL){
    	printf("can't open right\n");
    	exit(1);
    }

    FILE *backfd = fopen("./cube/back.rgba", "rb");
    if (backfd==NULL){
    	printf("can't open back\n");
    	exit(1);
    }

    FILE *topfd = fopen("./cube/top.rgba", "rb");
    if (topfd==NULL){
    	printf("can't open top\n");
    	exit(1);
    }

    FILE *bottomfd = fopen("./cube/bottom.rgba", "rb");
    if (bottomfd==NULL){
    	printf("can't open back\n");
    	exit(1);
    }


	FILE *planefd = fopen("plane.rgb", "wb+");
	if (planefd==NULL){
		printf("cant create output file\n");
		exit(1);
	}
	
	HANDLE_ERROR(hipHostAlloc((void**) &xymap, sizeof(int4) * OUT_X*OUT_Y,hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**) &bmapg, sizeof(float4) * OUT_X*OUT_Y,hipHostMallocDefault));

	HANDLE_ERROR(hipHostAlloc((void**) &plane, 4 * DEST_X*DEST_Y,hipHostMallocDefault));
	
	for(i=0;i<6;i++){
		HANDLE_ERROR(hipHostAlloc((void**) &sdata[i], 4 * SOURCE_X*SOURCE_Y,hipHostMallocDefault));
	}
	
	fread(xymap, sizeof(int4), OUT_X*OUT_Y, xymapfd);
	fread(bmapg, sizeof(float4), OUT_Y*OUT_X, bmapfd);
	fread(sdata[0], 4, SOURCE_X*SOURCE_Y, rightfd);
	
	fread(sdata[1], 4, SOURCE_X*SOURCE_Y, frfd);
	fread(sdata[2], 4, SOURCE_X*SOURCE_Y, leftfd);
	fread(sdata[3], 4, SOURCE_X*SOURCE_Y, backfd);
	fread(sdata[4], 4, SOURCE_X*SOURCE_Y, topfd);
	fread(sdata[5], 4, SOURCE_X*SOURCE_Y, bottomfd);
		
	fflush(rightfd);
	fflush(frfd);
	fflush(leftfd);
	fflush(backfd);
	fflush(topfd);
	fflush(bottomfd);

	fclose(rightfd);
	fclose(frfd);
	fclose(leftfd);
	fclose(backfd);
	fclose(topfd);
	fclose(bottomfd);

	float *dev_nv_wm;
	int4 *dev_xymap;
	float4 *dev_bmap;
	unsigned int *dev_source[6];
	unsigned int *dev_plane;
	
    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_nv_wm, sizeof(nv_wm) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_xymap, sizeof(int4)*OUT_X*OUT_Y ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_bmap,  sizeof(float4)*OUT_X*OUT_Y ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_plane, 4*DEST_Y*DEST_X ) );

    
    for (i=0;i<6;i++)
    	HANDLE_ERROR( hipMalloc( (void**)&dev_source[i], 4 * (SOURCE_X*SOURCE_Y) ) );
 	
 	printf("size of nv_wm: %lu\n", sizeof(nv_wm) );
    HANDLE_ERROR( hipMemcpy( dev_nv_wm, nv_wm, sizeof(nv_wm), hipMemcpyHostToDevice ) );
    printf("size of xymap: %lu\n", sizeof(xymap) );
    HANDLE_ERROR( hipMemcpy( dev_xymap, xymap, sizeof(int4)*OUT_X*OUT_Y, hipMemcpyHostToDevice ) );
    printf("size of bmapg: %lu\n", sizeof(bmapg) );
    HANDLE_ERROR( hipMemcpy( dev_bmap, bmapg, sizeof(float4)*OUT_X*OUT_Y, hipMemcpyHostToDevice ) );

hipEvent_t start1,start2, start3, stop1,stop2,stop3;
hipEventCreate(&start1);
hipEventCreate(&start2);
hipEventCreate(&start3);


hipEventCreate(&stop1);
hipEventCreate(&stop2);
hipEventCreate(&stop3);
int il;
for(il=0;il<20;il++){

hipEventRecord(start1);
    for(i=0;i<6;i++){
       	HANDLE_ERROR( hipMemcpy( dev_source[i], sdata[i], 4*SOURCE_X*SOURCE_Y, hipMemcpyHostToDevice ) );
    }
        
    hipEventRecord(stop1);
    dim3 grid(DEST_X/8,DEST_Y/8);
    dim3 block(8,8);

    hipEventRecord(start2);
    create_pano<<<grid,block>>>(	dev_nv_wm,
    							dev_xymap,
    							dev_bmap,
    							dev_source[0],
    							dev_source[1],
    							dev_source[2],
    							dev_source[3],
    							dev_source[4],
    							dev_source[5],
    							dev_plane
    							);
    hipEventRecord(stop2);

    hipEventRecord(start3);
    HANDLE_ERROR(hipMemcpy( plane, dev_plane, 4*DEST_Y*DEST_X, hipMemcpyDeviceToHost )); 
    hipEventRecord(stop3);
   

hipEventSynchronize(stop1);
float milliseconds = 0;
hipEventElapsedTime(&milliseconds, start1, stop1);
printf("sources copy time: %f\n",milliseconds );

hipEventSynchronize(stop2);
float milliseconds2 = 0;
hipEventElapsedTime(&milliseconds2, start2, stop2);
printf("kernel execution time: %f\n",milliseconds2 );

hipEventSynchronize(stop3);
float milliseconds3 = 0;
hipEventElapsedTime(&milliseconds3, start3, stop3);
printf("result copy time: %f\n",milliseconds3 );

}

	fwrite(plane, DEST_Y*DEST_X,4,planefd);
	fflush(planefd);
	fflush(xymapfd);
	fflush(bmapfd);
	fclose(planefd);
	fclose(xymapfd);
	fclose(bmapfd);

}


